#include "hip/hip_runtime.h"
// High-Performance Integrated hydrodynamic Modelling System ***hybrid***
// @author: Jiaheng Zhao (Hemlab)
// @license: (C) Copyright 2020-2025. 2025~ Apache Licence 2.0
// @contact: j.zhao@lboro.ac.uk
// @software: hipims_hybrid
// @time: 07.01.2021
// This is a beta version inhouse code of Hemlab used for high-performance flooding simulation.
// Feel free to use and extend if you are a ***member of hemlab***.
// #include <torch/extension.h>
#include "gpu.cuh"
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <torch/extension.h>

namespace
{
}

template <typename scalar_t>
__global__ void frictionCalculation_kernel(
    int N, int32_t *__restrict__ wetMask, scalar_t *__restrict__ h_update,
    scalar_t *__restrict__ qx_update, scalar_t *__restrict__ qy_update,
    scalar_t *__restrict__ z_update, uint8_t *__restrict__ landuse,
    scalar_t *__restrict__ h, scalar_t *__restrict__ wl,
    scalar_t *__restrict__ qx, scalar_t *__restrict__ qy,
    scalar_t *__restrict__ z, scalar_t *__restrict__ manning,
    scalar_t *__restrict__ dt)
{

  scalar_t h_small = 1.0e-6;
  scalar_t g = 9.81;
  scalar_t q_norm, C_f;
  // scalar_t C_f_temp;

  // get the index of cell
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (j < N)
  {
    int32_t i = wetMask[j];
    // first, we will update the water depth
    h[i] += h_update[i];
    if (h[i] < h_small)
    {
      qx[i] = 0.0;
      qy[i] = 0.0;
    }
    else
    {

      C_f =
          g * manning[landuse[i]] * manning[landuse[i]] * pow(h[i], -1.0 / 3.0);

      // increased manning coefficient
      // C_f_temp = g * min(manning[landuse[i]] * manning[landuse[i]] * 4.0, 0.36) * pow(h[i], -1.0 / 3.0);

      qx[i] += qx_update[i];
      qy[i] += qy_update[i];
      q_norm = sqrt(qx[i] * qx[i] + qy[i] * qy[i]);
      if (manning[landuse[i]] > 0.0)
      {
        // if (q_norm > 1.0e-10){
        if (abs(qx[i]) > 1.0e-10)
        {
          // if (C_f > 1.0e-15)

          // qx[i] = (1.0 -
          //          sqrt(1.0 + (4.0 * dt[0] * C_f) / (h[i] * h[i]) * q_norm)) /
          //         (-2.0 * dt[0] * C_f / (h[i] * h[i]) * (q_norm / qx[i]));

          auto temp = (1.0 -
                       sqrt(1.0 + (4.0 * dt[0] * C_f) / (h[i] * h[i]) * q_norm)) /
                      (-2.0 * dt[0] * C_f / (h[i] * h[i]) * (q_norm / qx[i]));

          // ==================================================//
          // add a limit for the supercritical flow
          // if fr>5.0, increase the manning value
          if (abs(temp) > 5.0 * h[i] * sqrt(h[i] * g))
          // if (abs(temp) > 10.0 * h[i])
          {
            qx[i] = 2.0 * temp - qx[i];
            // make sure the direction is the same with temp
            if (qx[i] * temp <= 0.0)
            {
              qx[i] = 0.0;
            }
            // qx[i] = (1.0 -
            //          sqrt(1.0 + (4.0 * dt[0] * C_f_temp) / (h[i] * h[i]) * q_norm)) /
            //         (-2.0 * dt[0] * C_f_temp / (h[i] * h[i]) * (q_norm / qx[i]));
          }
          else
          {
            qx[i] = temp;
          }
          // ==================================================//
        }
        if (abs(qy[i]) > 1.0e-10)
        {
          // if (C_f > 1.0e-15)

          // qy[i] = (1.0 -
          //          sqrt(1.0 + (4.0 * dt[0] * C_f) / (h[i] * h[i]) * q_norm)) /
          //         (-2.0 * dt[0] * C_f / (h[i] * h[i]) * (q_norm / qy[i]));
          auto temp = (1.0 -
                       sqrt(1.0 + (4.0 * dt[0] * C_f) / (h[i] * h[i]) * q_norm)) /
                      (-2.0 * dt[0] * C_f / (h[i] * h[i]) * (q_norm / qy[i]));

          // ==================================================//
          // add a limit for the supercritical flow
          // if fr>5.0, increase the manning value

          if (abs(temp) > 5.0 * h[i] * sqrt(h[i] * g))
          // if (abs(temp) > 10.0 * h[i])
          {
            qy[i] = 2.0 * temp - qy[i];
            // make sure the direction is the same with temp
            if (qy[i] * temp <= 0.0)
            {
              qy[i] = 0.0;
            }

            // qy[i] = (1.0 -
            //          sqrt(1.0 + (4.0 * dt[0] * C_f_temp) / (h[i] * h[i]) * q_norm)) /
            //         (-2.0 * dt[0] * C_f_temp / (h[i] * h[i]) * (q_norm / qy[i]));
          }
          else
          {
            qy[i] = temp;
          }
          // ==================================================//
        }
        // }
      }
    }
    // h[i] += h_update[i];
    z[i] += z_update[i];
    wl[i] = z[i] + h[i];

    h_update[i] = 0.0;
    qx_update[i] = 0.0;
    qy_update[i] = 0.0;
    z_update[i] = 0.0;
  }
}

void frictionCalculation_cuda(at::Tensor wetMask, at::Tensor h_update,
                              at::Tensor qx_update, at::Tensor qy_update,
                              at::Tensor z_update, at::Tensor landuse,
                              at::Tensor h, at::Tensor wl, at::Tensor qx,
                              at::Tensor qy, at::Tensor z, at::Tensor manning,
                              at::Tensor dt)
{
  const int N = wetMask.numel();
  at::cuda::CUDAGuard device_guard(h.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES(
      h.type(), "frictioncuda_Calculation", ([&] {
        frictionCalculation_kernel<
            scalar_t><<<GET_BLOCKS(N), CUDA_NUM_THREADS, 0, stream>>>(
            N, wetMask.data<int32_t>(), h_update.data<scalar_t>(),
            qx_update.data<scalar_t>(), qy_update.data<scalar_t>(),
            z_update.data<scalar_t>(), landuse.data<uint8_t>(),
            h.data<scalar_t>(), wl.data<scalar_t>(), qx.data<scalar_t>(),
            qy.data<scalar_t>(), z.data<scalar_t>(), manning.data<scalar_t>(),
            dt.data<scalar_t>());
      }));
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    printf("Error in load_textures: %s\n", hipGetErrorString(err));
}
